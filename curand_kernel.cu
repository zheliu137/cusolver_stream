#include <cstdio>
#include <cstdlib>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hiprand.h>
//#include "cuda_settings.h"

void createRandoms(int size, double *h_randomArray){
    hiprandGenerator_t generator;
    double *randomArray;
    hipMalloc((void**)&randomArray, size*size*sizeof(double));
    // hipHostMalloc((void**)&h_randomArray, size*size*sizeof(double));
    hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(generator,(int)time(NULL));
    hiprandGenerateUniformDouble(generator,randomArray,size*size);
    hipMemcpy(h_randomArray, randomArray, sizeof(double) * size * size, hipMemcpyDeviceToHost);
}
